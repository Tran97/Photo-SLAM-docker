#include <hip/hip_runtime.h>
#include <iostream>
#include <thread>
#include <chrono>

void monitorCudaMemory(int interval_ms) {
    size_t free_mem, total_mem;

    while (true) {
        // Query GPU memory usage
        hipError_t err = hipMemGetInfo(&free_mem, &total_mem);
        if (err != hipSuccess) {
            std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
            return;
        }

        size_t used_mem = total_mem - free_mem;

        // Display memory usage
        std::cout << "Used Memory: " << used_mem / (1024.0 * 1024.0) << " MB"
                  << " / Total Memory: " << total_mem / (1024.0 * 1024.0) << " MB" << std::endl;

        // Wait for the specified interval
        std::this_thread::sleep_for(std::chrono::milliseconds(interval_ms));
    }
}

int main() {
    int interval_ms = 1000; // Update interval in milliseconds

    std::cout << "Monitoring GPU memory usage. Press Ctrl+C to stop." << std::endl;
    monitorCudaMemory(interval_ms);

    return 0;
}
